#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <mpi.h>

MPI_Status status;

__global__ void matrixMul(float* MatA, float* MatB, float* MatC, int arr_size, int my_rank, int node_num)
{
	int i = threadIdx.x;
	int j = blockIdx.x;	
	int start_range = (arr_size/node_num)*(my_rank);
        int end_range = ((my_rank+1)*(arr_size/node_num));
	
	if(start_range<=j && j<end_range)
	{
		for(int x=0 ;x<arr_size ; x++)
		{	
			MatC[arr_size*j + i] += MatA[arr_size*j + x] * MatB[arr_size * x + i];
		}
	}	
}

int main(int argc, char** argv)
{
	int n = 1024;
	
	int size, myrank;

	float* host_MatA;
	float* host_MatB;
	float* host_MatC;
	float* host_tmp;	

	float* dev_MatA;
	float* dev_MatB;
	float* dev_MatC;

	size_t bytes = n * n * sizeof(float);

	MPI_Init(&argc, &argv);
	MPI_Comm_rank(MPI_COMM_WORLD, &myrank);
	MPI_Comm_size(MPI_COMM_WORLD, &size);

	host_MatA = (float*)malloc(bytes);
	host_MatB = (float*)malloc(bytes);
	host_MatC = (float*)malloc(bytes);
	host_tmp = (float*)malloc(bytes);

	for(int i = 0; i < n; i++)
        {
        	for(int j = 0; j < n; j++)
                {
                         host_MatA[i * n + j] = 1;//rand() % 32;
                         host_MatB[i * n + j] = 1;//rand() % 32;
			 host_MatC[i * n + j] = 0;
			 host_tmp[i * n + j] = 0;
                }
        }

	hipMalloc((void**)&dev_MatA, bytes);
	hipMalloc((void**)&dev_MatB, bytes);
	hipMalloc((void**)&dev_MatC, bytes);
       	
	if(myrank == 0)
        {
		for(int i=1; i<size; i++)
		{
			MPI_Send(host_MatA, n*n, MPI_FLOAT, i, 1, MPI_COMM_WORLD);
	        	MPI_Send(host_MatB, n*n, MPI_FLOAT, i, 1, MPI_COMM_WORLD);
                        MPI_Send(host_MatC, n*n, MPI_FLOAT, i, 1, MPI_COMM_WORLD);
		}

		hipMemcpy(dev_MatA, host_MatA, bytes, hipMemcpyHostToDevice);
                hipMemcpy(dev_MatB, host_MatB, bytes, hipMemcpyHostToDevice);
                hipMemcpy(dev_MatC, host_MatC, bytes, hipMemcpyHostToDevice);
		
		matrixMul<<<n, n>>>(dev_MatA, dev_MatB, dev_MatC, n, myrank, size);
		hipDeviceSynchronize();
		hipMemcpy(host_MatC, dev_MatC, bytes, hipMemcpyDeviceToHost);
		
		for(int i=1; i<size ; i++)
		{
			MPI_Recv(host_tmp, n*n, MPI_FLOAT, i, 1, MPI_COMM_WORLD, &status);
	                for(int i = 0; i < n; i++)
        	        {
                	         for(int j = 0; j < n; j++)
                       		 {
                                	host_MatC[i * n + j] = host_MatC[i * n + j] + host_tmp[i * n + j];
                        	 }
                	}
		}
        }
	else if(myrank > 0)
        {
	        float* slave_MatA = (float*)malloc(bytes);
        	float* slave_MatB = (float*)malloc(bytes);
	        float* slave_MatC = (float*)malloc(bytes);
	
                MPI_Recv(slave_MatA, n*n, MPI_FLOAT, 0, 1, MPI_COMM_WORLD, &status);
		MPI_Recv(slave_MatB, n*n, MPI_FLOAT, 0, 1, MPI_COMM_WORLD, &status);
		MPI_Recv(slave_MatC, n*n, MPI_FLOAT, 0, 1, MPI_COMM_WORLD, &status);
		
                hipMemcpy(dev_MatA, slave_MatA, bytes, hipMemcpyHostToDevice);
                hipMemcpy(dev_MatB, slave_MatB, bytes, hipMemcpyHostToDevice);
                hipMemcpy(dev_MatC, slave_MatC, bytes, hipMemcpyHostToDevice);

		matrixMul<<<n, n>>>(dev_MatA, dev_MatB, dev_MatC, n, myrank, size);
		hipDeviceSynchronize();
		hipMemcpy(slave_MatC, dev_MatC, bytes, hipMemcpyDeviceToHost);
				
		MPI_Send(slave_MatC, n*n, MPI_FLOAT, 0, 1, MPI_COMM_WORLD);
	        free(slave_MatA);
       		free(slave_MatB);
        	free(slave_MatC);
        }
	hipDeviceSynchronize();
	if(myrank == 0)
	{
		for(int i = 0; i < n*n; i++)
        	{
                	if(i%n == 0) printf("\n");
                	printf("[%d]%.1f ",i, host_MatC[i]);
        	}
	}
	
	free(host_MatA);
	free(host_MatB);
	free(host_MatC);
	free(host_tmp);

	hipFree(dev_MatA);
	hipFree(dev_MatB);
	hipFree(dev_MatC);
	
	MPI_Finalize();
	return 0;
}
