#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<iostream>
#include<algorithm>
#include<time.h>
#include<hip/hip_runtime.h>
#include<mpi.h>

using namespace std;

MPI_Status status;

__global__ void avg_pooling(float* gpu_input, float* gpu_output_data, int input_h_size, int input_w_size, int pool_h_size, int pool_w_size, int pool_h_stride, int pool_w_stride, int start, int end) 
{
    int x = blockIdx.x;
    int y = blockIdx.y;

    if(start <= y && y < end)
    {
        int sum;
        float avg;
        
        int pooled_size = ((input_w_size - pool_w_size) / pool_w_stride) + 1;

	    int h_start = y * pool_h_stride;
        int w_start = x * pool_w_stride;
        int h_end = min(h_start + pool_h_size, input_h_size);
        int w_end = min(w_start + pool_w_size, input_w_size);

        h_start = max(h_start, 0);
        w_start = max(w_start, 0);
        sum = 0;
        avg = 0;

        int pool_index = (y * pooled_size) + x;
        for (int h = h_start; h < h_end; h++)
        {
            for (int w = w_start; w < w_end; w++)
            {
                int index = (h * input_w_size) + w;
                sum += gpu_input[index];
            }
            avg = (float)sum / (pool_h_size * pool_w_size);
            gpu_output_data[pool_index] = avg;
        }
    }
}

void Init_input(float* input, int input_h_size, int input_w_size, int num)
{
        srand(time(NULL));

        for (int h = 0; h < input_h_size; h++)
        {
        	for (int w = 0; w < input_w_size; w++)
                {
                	input[(h * input_w_size) + w] = rand() % num;
                }
        }

}

void print(float* data, int h_size, int w_size)
{
	for (int h = 0; h < h_size; h++)
    {
        for (int w = 0; w < w_size; w++)
        {
           	printf("%.2f ", data[(h * w_size) + w]);
		}
	    printf("\n");
    }
    printf("\n");
}

int main(int argc, char** argv)
{
    int procs, myrank;
    int offset = 0;
    int before_offset = 0;

	int input_h_size = 8;
	int input_w_size = 8;
	int pool_w_size = 2;
    int pool_h_size = 2;
    int pool_w_stride = 2;
    int pool_h_stride = 2;
    
    int input_size = input_h_size * input_w_size;

	int pooled_h = ((input_h_size - pool_h_size) / pool_h_stride) + 1;
    int pooled_w = ((input_w_size - pool_w_size) / pool_w_stride) + 1;	

	float* input = (float*)malloc(sizeof(float) * input_size);
    float* result = (float*)malloc(sizeof(float) * input_size);
    float* host_tmp = (float*)malloc(sizeof(float) * input_size);
    float* slave_input = (float*)malloc(sizeof(float) * input_size);
    float* slave_result = (float*)malloc(sizeof(float) * pooled_h * pooled_w);

	float* gpu_output_data;
    float* gpu_input;
    
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &myrank);
    MPI_Comm_size(MPI_COMM_WORLD, &procs);

    Init_input(input, input_h_size, input_w_size, 10);
    /*초기값 출력*/
    if(myrank == 0)
    {
        printf("===초기화된 행렬 값===\n");
        print(input, input_h_size, input_w_size);
    }

	hipMalloc((void**)&gpu_input, sizeof(float) * input_size);
    hipMalloc((void**)&gpu_output_data, sizeof(float) * input_size);

    dim3 dimGrid(input_h_size, input_w_size);
    dim3 dimBlock(1, 1);
    
    if(myrank == 0)
    {
        int start = (input_size/procs)*myrank;
        int end = ((myrank+1)*(input_size/procs));

        for(int i = 1; i < procs; i++)
        {
            MPI_Send(input, input_size, MPI_FLOAT, i, 0, MPI_COMM_WORLD);
        }
        hipMemcpy(gpu_input, input, sizeof(float) * input_size, hipMemcpyHostToDevice);

	    avg_pooling<<<dimGrid,dimBlock>>>(gpu_input, gpu_output_data, input_h_size, input_w_size, pool_h_size, pool_w_size, pool_h_stride, pool_w_stride, start, end);
        hipDeviceSynchronize();

        hipMemcpy(result, gpu_output_data, sizeof(float) * input_size, hipMemcpyDeviceToHost);
        printf("=======rank = %d 계산된 값 ========\n\n",myrank);
        print(result,pooled_h, pooled_w);
        printf("=======end 값 ========\n\n");

        offset = (int)input_w_size / procs;
        for(int i = 1; i < procs; i++)
        {
            MPI_Recv(host_tmp, input_size, MPI_FLOAT, i, 1, MPI_COMM_WORLD, &status);
            before_offset = offset;
            offset += (pooled_h*pooled_w / procs);
            for(int h = before_offset; h < offset; h++)
            {
                for(int w = 0; w < input_h_size; w++)
                {
                    result[(h * input_h_size + w)] = result[(h * input_h_size + w)] + host_tmp[(h * input_h_size + w)];
                }
            }
        }
    }

    if(myrank > 0)
    {
        int start = ((input_size)/procs)*myrank;
        int end = ((myrank+1)*((input_size)/procs));

        float* slave_input = (float*)malloc(sizeof(float) * input_size);
        float* slave_result = (float*)malloc(sizeof(float) * input_size);

        MPI_Recv(slave_input, input_size, MPI_FLOAT, 0, 0, MPI_COMM_WORLD, &status);

        hipMemcpy(gpu_input, slave_input, sizeof(float) *input_size, hipMemcpyHostToDevice);

        dim3 dimGrid(input_h_size, input_w_size);
        avg_pooling<<<dimGrid,dimBlock>>>(gpu_input, gpu_output_data, input_h_size, input_w_size, pool_h_size, pool_w_size, pool_h_stride, pool_w_stride, start, end);
        hipDeviceSynchronize();
        
        hipMemcpy(slave_result, gpu_output_data, sizeof(float) * input_size, hipMemcpyDeviceToHost);
        printf("=======rank = %d 계산된 값 ========\n\n",myrank);
        print(slave_result,pooled_h, pooled_w);
        printf("=======end 값 ========\n\n");

        MPI_Send(slave_result, input_size, MPI_FLOAT, 0, 1, MPI_COMM_WORLD);

        free(slave_input);
        free(slave_result);
    }
    hipDeviceSynchronize();

    if(myrank == 0)
    {
        printf("----------------------\n\n");
        printf("===pooling된 행렬===\n");
	    print(result, pooled_h, pooled_w);
    }

    free(input);
    free(result);
    free(host_tmp);
    
	hipFree(gpu_output_data);
    hipFree(gpu_input);

    MPI_Finalize();
	return 0; 
}