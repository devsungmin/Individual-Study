#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <mpi.h>
#include <time.h>

MPI_Status status;

__global__ void matrixMul(float* MatA, float* MatB, float* MatC, int arr_size, int start_range, int end_range)
{
	int i = threadIdx.x;
	int j = blockIdx.x;	
	
	if(start_range<=j && j<end_range)
	{
		for(int x=0 ;x<arr_size ; x++)
		{	
			MatC[arr_size*j + i] += MatA[arr_size*j + x] * MatB[arr_size * x + i];
		}
	}	
}

int main(int argc, char** argv)
{
	int n = 1024;
	int offset=0;
	int before_offset=0;
	int size, myrank;

	float* host_MatA;
	float* host_MatB;
	float* host_MatC;
	float* host_tmp;	

	float* dev_MatA;
	float* dev_MatB;
	float* dev_MatC;

	size_t bytes = n * n * sizeof(float);
	
	clock_t start, end;
	float result = 0;
	
	MPI_Init(&argc, &argv);
	MPI_Comm_rank(MPI_COMM_WORLD, &myrank);
	MPI_Comm_size(MPI_COMM_WORLD, &size);

	host_MatA = (float*)malloc(bytes);
	host_MatB = (float*)malloc(bytes);
	host_MatC = (float*)malloc(bytes);
	host_tmp = (float*)malloc(bytes);

	for(int i = 0; i < n; i++)
        {
        	for(int j = 0; j < n; j++)
                {
                         host_MatA[i * n + j] = 1;//rand() % 32;
                         host_MatB[i * n + j] = 1;//rand() % 32;
			 host_MatC[i * n + j] = 0;
			 host_tmp[i * n + j] = 0;
                }
        }

	hipMalloc((void**)&dev_MatA, bytes);
	hipMalloc((void**)&dev_MatB, bytes);
	hipMalloc((void**)&dev_MatC, bytes);
       	
	start = clock();	

	if(myrank == 0)
        {
                int start_range = (n/size)*(myrank);
                int end_range = ((myrank+1)*(n/size));

		for(int i=1; i<size; i++)
		{
			MPI_Send(host_MatA, n*n, MPI_FLOAT, i, 1, MPI_COMM_WORLD);
	        	MPI_Send(host_MatB, n*n, MPI_FLOAT, i, 1, MPI_COMM_WORLD);
                        MPI_Send(host_MatC, n*n, MPI_FLOAT, i, 1, MPI_COMM_WORLD);
		}

		hipMemcpy(dev_MatA, host_MatA, bytes, hipMemcpyHostToDevice);
                hipMemcpy(dev_MatB, host_MatB, bytes, hipMemcpyHostToDevice);
                hipMemcpy(dev_MatC, host_MatC, bytes, hipMemcpyHostToDevice);
				
		matrixMul<<<n, n>>>(dev_MatA, dev_MatB, dev_MatC, n, start_range, end_range);
		hipDeviceSynchronize();
		hipMemcpy(host_MatC, dev_MatC, bytes, hipMemcpyDeviceToHost);
		
		offset = (int)n/size;
		for(int i=1; i<size ; i++)
		{
			MPI_Recv(host_tmp, n*n, MPI_FLOAT, i, 1, MPI_COMM_WORLD, &status);
			before_offset = offset;
			offset+=(n/size);
	                for(int i = before_offset; i < offset; i++)
        	        {
                	         for(int j = 0; j < n; j++)
                       		 {
                                	host_MatC[i * n + j] = host_MatC[i * n + j] + host_tmp[i * n + j];
                        	 }
                	}
		}
        }
	else if(myrank > 0)
        {
                int start_range = (n/size)*(myrank);
                int end_range = ((myrank+1)*(n/size));

	        float* slave_MatA = (float*)malloc(bytes);
        	float* slave_MatB = (float*)malloc(bytes);
	        float* slave_MatC = (float*)malloc(bytes);
	
                MPI_Recv(slave_MatA, n*n, MPI_FLOAT, 0, 1, MPI_COMM_WORLD, &status);
		MPI_Recv(slave_MatB, n*n, MPI_FLOAT, 0, 1, MPI_COMM_WORLD, &status);
		MPI_Recv(slave_MatC, n*n, MPI_FLOAT, 0, 1, MPI_COMM_WORLD, &status);
		
                hipMemcpy(dev_MatA, slave_MatA, bytes, hipMemcpyHostToDevice);
                hipMemcpy(dev_MatB, slave_MatB, bytes, hipMemcpyHostToDevice);
                hipMemcpy(dev_MatC, slave_MatC, bytes, hipMemcpyHostToDevice);

		matrixMul<<<n, n>>>(dev_MatA, dev_MatB, dev_MatC, n, start_range, end_range);
		hipDeviceSynchronize();
		hipMemcpy(slave_MatC, dev_MatC, bytes, hipMemcpyDeviceToHost);
				
		MPI_Send(slave_MatC, n*n, MPI_FLOAT, 0, 1, MPI_COMM_WORLD);
	        free(slave_MatA);
       		free(slave_MatB);
        	free(slave_MatC);
        }
	hipDeviceSynchronize();
	end = clock();
	result = (float)(end - start)/CLOCKS_PER_SEC;	
	/*
	if(myrank == 0)
	{
		for(int i = 0; i < n*n; i++)
        	{
                	if(i%n == 0) printf("\n");
                	printf("[%d]%.1f ",i, host_MatC[i]);
        	}
	}
	*/
	printf("rank : %d  time : %.4f\n", myrank, result);
	
	free(host_MatA);
	free(host_MatB);
	free(host_MatC);
	free(host_tmp);

	hipFree(dev_MatA);
	hipFree(dev_MatB);
	hipFree(dev_MatC);
	
	MPI_Finalize();
	return 0;
}
